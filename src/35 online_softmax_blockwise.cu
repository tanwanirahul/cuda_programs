#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "matrix_utils.h"
#include "timer.h"
#include <math.h>

#define EPS 0.00001
#define THREADS_PER_BLOCK 1024
#define MAX_SRAM_ELEMS 1024

#define NUM_ROWS 10240
#define NUM_COLS 10240


void softmax_CPU(float *mat, float *result, unsigned int M, unsigned int N) {

    for(unsigned int i = 0; i<M; i++) {
        float rowMax = -INFINITY;
        float rowSum = 0.0;

        // Find RowMax.
        for(unsigned int j = 0; j<N; j++) {
            float current = mat[(i * N) + j];
            if(current > rowMax)
                rowMax = current;
        }

        // Find Exponents and RowSum
        for(unsigned int j = 0; j<N; j++) {
            float current = mat[(i * N) + j];
            float elementExp = exp(current - rowMax);
            result[(i*N) + j] = elementExp;
            rowSum+= elementExp;
        }

        // Normalize by RowSum;
        rowSum = (rowSum == 0.0)? EPS : rowSum;
        for(unsigned int j = 0; j<N; j++)
            result[(i*N) + j] = result[(i*N) + j] / rowSum;
    }
}

/**
 * Online softmax improves upon the prev. softmax implementation using shared memory. 
 * Instead of loading the whole row of the input matrix into shared memory, online
 * softmax loads part of the row and computes the running max and sum. 
 * 
 * The final values are adjusted in the second pass after the global max has been identified.
 * 
 * In addition to being effective on large row data, online softmax has another advantage
 * that it only needs two loops/passes over the data rather than three loops/passes as in
 * standard softmax.
 * 
 * Given the M*N matrix mat, computes the softmax and stores the output
 * at the memory location pointed to by result.
 */
__global__ void softmax_kernel(float *mat, float *result, unsigned int M, unsigned int N) {

    // Each block is responsible for loading N elements.
    unsigned int rowStart = (blockIdx.x * N);

    // Shared memory to contain the row Data.
    __shared__ float rowData[MAX_SRAM_ELEMS];
    __shared__ float rowSum_s;
    __shared__ float rowMax_s;

    // Initialize shared memory.
    if(threadIdx.x == 0) {
        rowSum_s = 0.0;
        rowMax_s = -INFINITY;
    }

    __syncthreads();

     // We may need multiple passes for loading the entire row of data.
    unsigned int dataLoadPasses = (N + MAX_SRAM_ELEMS - 1) / MAX_SRAM_ELEMS;
    for(unsigned int dPass=0; dPass < dataLoadPasses; dPass++) {

        // Load the tile of row data into shared memory.
        for(unsigned int i=0; i<(MAX_SRAM_ELEMS + blockDim.x - 1) / blockDim.x; i++)
        {
            unsigned int elemIdx = (i * blockDim.x) + threadIdx.x;
            if(elemIdx < MAX_SRAM_ELEMS) {
                    if(((dPass * MAX_SRAM_ELEMS) + elemIdx) < N) {
                        rowData[elemIdx] = mat[rowStart + (dPass * MAX_SRAM_ELEMS) + elemIdx];
                    } else {
                        rowData[elemIdx] = 0.0;
                    }
            }
        }
        __syncthreads();

        // We need to compute the running Max and Sum for the loaded tile of data using
        // a single thread.
        if(threadIdx.x == 0)
        {
            float prevMax =  rowMax_s;
            float rowSum = rowSum_s;

            for(unsigned int i=0; i<MAX_SRAM_ELEMS; i++) {
                float currMax = (prevMax < rowData[i])? rowData[i] : prevMax;
                rowSum = rowSum * exp(prevMax - currMax) + exp(rowData[i] - currMax);
                prevMax = currMax;
            }

            rowSum_s = rowSum;
            rowMax_s = prevMax;
        }        
        __syncthreads();

    }
       
    rowSum_s = (rowSum_s == 0.0)? EPS : rowSum_s;

    // Second Loop/Pass - Normalize the data using the computed rowSum and global Max;
    for(unsigned int i=0; i < (N + blockDim.x - 1) / blockDim.x; i++) {

        if( ((i*blockDim.x) + threadIdx.x) < N)
        {
            float inputElem = mat[rowStart + (i * blockDim.x) + threadIdx.x];
            result[rowStart + (i * blockDim.x) + threadIdx.x] = exp(inputElem - rowMax_s) / (rowSum_s);
        }
    }
   
}

void softmax_wrapper(float *mat, float * result, unsigned int M, unsigned int N) {
    // Allocate memory on the GPU device.
    float *mat_d, *result_d;
    size_t mat_size = sizeof(float) * M * N;

    Timer timer;
    timer = initTimer(1);
    startTimer(&timer);

    hipError_t error;
    error = hipMalloc((void **) &mat_d, mat_size);
    if (error != hipSuccess) {
        printf("\nfailed to allocate memory on CUDA device for input matrix.\n");
        return;
    }
    error = hipMalloc((void **) &result_d, mat_size);
    if (error != hipSuccess) {
        printf("\nfailed to allocate memory on CUDA device for result matrix.\n");
        return;
    }
    hipDeviceSynchronize();
    printf("Allocated required memory on the CUDA device.\n\n");
    stopAndPrintElapsed(&timer, "GPU Device Memory Allocation Time: ", CYAN);


    // Copy data from Host to GPU.
    timer = initTimer(1);
    startTimer(&timer);

    error = hipMemcpy(mat_d, mat, mat_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("\nfailed to copy input matrix on CUDA device.");
        return;
    }
    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "Time To Copy Data to GPU DRAM: ", CYAN);


    // Do the computation on the device.
    timer = initTimer(1);
    startTimer(&timer);

    unsigned int threadsPerBlock = (THREADS_PER_BLOCK < N)? THREADS_PER_BLOCK: N;
    unsigned int numBlocks = M;
    
    softmax_kernel <<<numBlocks, threadsPerBlock>>>(mat_d, result_d, M, N);

    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "CUDA Kernel Execution Time: ", GREEN);

    // Copy the results back from GPU  to Host.
    timer = initTimer(1);
    startTimer(&timer);

    hipMemcpy(result, result_d, mat_size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "Time to COPY Results from GPU to HOST: ", CYAN);

    // Deallocate memory from the GPU device.
    hipFree((void*) mat_d);
    hipFree((void*) result_d);
    hipDeviceSynchronize();

}

int main(int argc, char** argv) {

    //unsigned int N = 40000;
    unsigned int N = NUM_COLS;
    unsigned int M = NUM_ROWS;

    size_t mat_size = sizeof(float) * M * N;

    // A - N*N matrix with Random values clipped by 10
    Matrix A = random_clipped_matrix_2D(M , N, 10);

    printf("Initialized an input matrix\n");

    // Allocate memory on the host for holding resulting matrix.
    float * result_CPU = (float*) malloc(mat_size);
    float * result_GPU = (float*) malloc(mat_size);

    Timer timer;
    timer = initTimer(1);
    startTimer(&timer);
    softmax_CPU(A.buffer, result_CPU, M, N);
    stopAndPrintElapsed(&timer, "CPU Execution Time: ", CYAN);


    // Convert the image to grey;
    timer = initTimer(1);
    startTimer(&timer);
    softmax_wrapper(A.buffer, result_GPU, M, N);
    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "GPU Execution Time: ", GREEN);

    Matrix cpuResult, gpuResult;
    cpuResult.rows = M;
    cpuResult.cols = N;
    cpuResult.buffer = result_CPU;

    gpuResult.rows = M;
    gpuResult.cols = N;
    gpuResult.buffer = result_GPU;

    bool areEqual = are_matrix_close(&cpuResult, &gpuResult, 0.000001f);
    printf("\nDo results from CPU and GPU implementation match? %d\n", areEqual);

    // Manually analyze the first few elements of the matrix to compare results
    int limit = (N < 1024)? N : 1024;
    printf("\nResults of Softmax for a row (Input   CPU     GPU): \n");
    for(int i =0; i<limit; i++) {
      printf("%d: %20.6f  %20.6f  %20.6f\n", i, A.buffer[i], result_CPU[i], result_GPU[i]);
    }

    // Free up allocated space on the host Matrix A, B and buffer holding the results.
    release_matrix(&A);
    free(result_CPU);
    free(result_GPU);

    return 0;
}