#include "hip/hip_runtime.h"

#include "timer.h"

# define BLOCK_SIZE 1024
/**
 * Implements the Scan part of the Kogge Scone algorithm. Kogge Scone algorithm 
 * is implemented using two kernels  - scan and add. 
 */
__global__ void scan_kernel(float *input, float * output, float *block_wise_sums, unsigned int N) {

    unsigned int seg_start = blockIdx.x * blockDim.x * 2;

    __shared__ float input_s[BLOCK_SIZE * 2];

    // Load input data into the shared memory.
    if(seg_start + threadIdx.x < N) {
        input_s[threadIdx.x] = input[seg_start + threadIdx.x];
    }
    else {
        input_s[threadIdx.x] = 0.0;
    }
    if(seg_start + threadIdx.x + BLOCK_SIZE < N) {
        input_s[threadIdx.x + BLOCK_SIZE] = input[seg_start + threadIdx.x + BLOCK_SIZE];
    }
    else {
        input_s[threadIdx.x + BLOCK_SIZE] = 0.0;
    }
    __syncthreads();


    // Reduction Step.
    for(int step =1; step<=BLOCK_SIZE; step*=2 ) {
        unsigned int i = (threadIdx.x + 1) * 2 * step -1;
        if(i < 2 * BLOCK_SIZE) {
            input_s[i] += input_s[i - step];
        }
        __syncthreads();
    }

    // Post Reduction Step.
    for(int step = BLOCK_SIZE/2; step>=1; step/=2) {
        unsigned int i = (threadIdx.x + 1) * 2 * step -1;
        if(i + step < 2 * BLOCK_SIZE) {
            input_s[i + step] += input_s[i];
        }
        __syncthreads();
    }

    // Store the block-wise partial sums.
    if(threadIdx.x == 0) {
        block_wise_sums[blockIdx.x] = input_s[2 * BLOCK_SIZE - 1];
    }

    // Update the sums in the output array in Global memory from the shared memory.
    // Load input data into the shared memory.
    if(seg_start + threadIdx.x < N) {
        output[seg_start + threadIdx.x] = input_s[threadIdx.x];
    }
    if(seg_start + threadIdx.x + BLOCK_SIZE < N) {
        output[seg_start + threadIdx.x + BLOCK_SIZE] = input_s[threadIdx.x + BLOCK_SIZE];
    }

}

/**
 * Adds total sum upto the prev block to every element in the array
 * except for elements in the first block.
 */
__global__ void add_kernel(float * output, float * block_wise_sums, unsigned int N) {
    unsigned int i = (blockIdx.x * blockDim.x * 2) + threadIdx.x;
    // Updathe output with partial sums.
    if(blockIdx.x > 0) {
        if (i < N)
            output[i] += block_wise_sums[blockIdx.x - 1];
        if (i + BLOCK_SIZE < N)
            output[i + BLOCK_SIZE] += block_wise_sums[blockIdx.x - 1];

    }
}

void scan_wrapper_d(float *input_d, float * output_d, unsigned int N) {
    unsigned int threadsPerBlock = BLOCK_SIZE;
    unsigned int numBlocks = (N + (2 * threadsPerBlock) - 1) / (2 * threadsPerBlock);

    float * blockWiseSums_d;
    hipMalloc((void**) &blockWiseSums_d, numBlocks * sizeof(float) );
    hipDeviceSynchronize();

    scan_kernel<<<numBlocks, threadsPerBlock>>>(input_d, output_d, blockWiseSums_d, N);
    hipDeviceSynchronize();

    // Perform scan on the block wise sum and then add the sums to output elements.
    if (numBlocks > 1) {
        scan_wrapper_d(blockWiseSums_d, blockWiseSums_d, numBlocks);
        add_kernel<<<numBlocks, threadsPerBlock>>>(output_d, blockWiseSums_d, N);
    }
    hipDeviceSynchronize();

    hipFree((void*)blockWiseSums_d);
    hipDeviceSynchronize();
}

void scan_wrapper(float *input, float *output, unsigned int N)
{
    // Step 1 - Allocate memory of the GPU device.
    float *input_d, *output_d;

    Timer timer;
    timer = initTimer(1);
    startTimer(&timer);
    hipMalloc((void**) &input_d, N * sizeof(float));
    hipMalloc((void**) &output_d, N * sizeof(float));

    // Copy data from host to GPU device.
    hipMemcpy(input_d, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTimer(&timer);
    printElapsedTime(&timer, "Data Transfer from Host to GPU:", CYAN);
    
    
    // Perform parallel computation.
    timer = initTimer(1);
    startTimer(&timer);
    scan_wrapper_d(input_d, output_d, N);
    hipDeviceSynchronize();
    stopTimer(&timer);
    printElapsedTime(&timer, "GPU Kernel Execution Time:", GREEN);

    // Copy results from GPU device to host memory.
    timer = initTimer(1);
    startTimer(&timer);
    hipMemcpy(output, output_d, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTimer(&timer);
    printElapsedTime(&timer, "Time to Copy Output from GPU to Host:", CYAN);

    // Deallocate memory on GPU device.
    hipFree((void*)input_d);
    hipFree((void*)output_d);
    hipDeviceSynchronize();

}

int main(int argc, char **argv)
{
    unsigned int N = (argc > 1)?(atoi(argv[1])):(1 << 30);
    
    // Allocate memory on the host.
    float *input = (float*) malloc(N * sizeof(float));
    float *output = (float*) malloc(N * sizeof(float));

    // create a random data for addition.
    for (unsigned int i = 0; i < N; i++) {
        //input[i] = rand();
        input[i] = 1.0;
    }

    Timer timer;
    //Timer timer_gpu;
    timer = initTimer(1);
    startTimer(&timer);
    scan_wrapper(input, output, N);
    stopAndPrintElapsed(&timer, "GPU End to End Executiom Time: ", GREEN);

    float sum = output[N-1];
    printf("No. of elements: %u\n", N);
    printf("Sum returned: %20.2f\n", sum);
    printf("Exepcted answer: %20.2f\n",N * 1.0 ); 
    printf("Does output match expected value? %d\n", (sum==(N * 1.0))?1:0);

    free(input);
    free(output);

    return 0;
}
