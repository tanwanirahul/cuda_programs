#include "hip/hip_runtime.h"
#include "timer.h"

# define BLOCK_SIZE 1024
/**
 * Implements the Scan part of the Kogge Scone algorithm. Kogge Scone algorithm 
 * is implemented using two kernels  - scan and add. 
 */
__global__ void scan_kernel(float *input, float * output, float *block_wise_sums, unsigned int N) {

    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i < N) {
        output[i] = input[i];
        __syncthreads();

        for(int step = 1; step <= BLOCK_SIZE/2; step*=2) {
            float sum = 0.0;
            if(threadIdx.x >= step) {
                sum = output[i] + output[i - step];
            }
            __syncthreads();
            if(threadIdx.x >= step) {
                output[i] = sum;
            }
            __syncthreads();
        }
        if(threadIdx.x == BLOCK_SIZE - 1 || i == (N-1)) {
            block_wise_sums[blockIdx.x] = output[i];
        }
    }
}

/**
 * Adds total sum upto the prev block to every element in the array
 * except for elements in the first block.
 */
__global__ void add_kernel(float * output, float * block_wise_sums, unsigned int N) {
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < N) {
        if(blockIdx.x > 0) {
            output[i] += block_wise_sums[blockIdx.x - 1];
        }
    }
}

void scan_wrapper_d(float *input_d, float * output_d, unsigned int N) {
    unsigned int threadsPerBlock = BLOCK_SIZE;
    unsigned int numBlocks = (N + (threadsPerBlock) - 1) / (threadsPerBlock);

    float * blockWiseSums_d;
    hipMalloc((void**) &blockWiseSums_d, numBlocks * sizeof(float) );
    hipDeviceSynchronize();

    scan_kernel<<<numBlocks, threadsPerBlock>>>(input_d, output_d, blockWiseSums_d, N);
    hipDeviceSynchronize();

    float * blockWiseSums_CPU = (float *) malloc(sizeof(float) * numBlocks);
    hipMemcpy(blockWiseSums_CPU, blockWiseSums_d, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    // Perform scan on the block wise sum and then add the sums to output elements.
    if (numBlocks > 1) {
        scan_wrapper_d(blockWiseSums_d, blockWiseSums_d, numBlocks);
        add_kernel<<<numBlocks, threadsPerBlock>>>(output_d, blockWiseSums_d, N);
    }
    hipDeviceSynchronize();

    hipFree((void*)blockWiseSums_d);
    hipDeviceSynchronize();
}

void scan_wrapper(float *input, float *output, unsigned int N)
{
    // Step 1 - Allocate memory of the GPU device.
    float *input_d, *output_d;

    Timer timer;
    timer = initTimer(1);
    startTimer(&timer);
    hipMalloc((void**) &input_d, N * sizeof(float));
    hipMalloc((void**) &output_d, N * sizeof(float));

    // Copy data from host to GPU device.
    hipMemcpy(input_d, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTimer(&timer);
    printElapsedTime(&timer, "Data Transfer from Host to GPU:", CYAN);
    
    
    // Perform parallel computation.
    timer = initTimer(1);
    startTimer(&timer);
    scan_wrapper_d(input_d, output_d, N);
    hipDeviceSynchronize();
    stopTimer(&timer);
    printElapsedTime(&timer, "GPU Kernel Execution Time:", GREEN);

    // Copy results from GPU device to host memory.
    timer = initTimer(1);
    startTimer(&timer);
    hipMemcpy(output, output_d, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTimer(&timer);
    printElapsedTime(&timer, "Time to Copy Output from GPU to Host:", CYAN);

    // Deallocate memory on GPU device.
    hipFree((void*)input_d);
    hipFree((void*)output_d);
    hipDeviceSynchronize();

}

int main(int argc, char **argv)
{
    unsigned int N = (argc > 1)?(atoi(argv[1])):(1 << 30);
    
    // Allocate memory on the host.
    float *input = (float*) malloc(N * sizeof(float));
    float *output = (float*) malloc(N * sizeof(float));

    // create a random data for addition.
    for (unsigned int i = 0; i < N; i++) {
        //input[i] = rand();
        input[i] = i;
    }

    Timer timer;
    //Timer timer_gpu;
    timer = initTimer(1);
    startTimer(&timer);
    scan_wrapper(input, output, N);
    stopAndPrintElapsed(&timer, "GPU End to End Executiom Time: ", GREEN);

    float sum = output[N-1];
    printf("No. of elements: %u\n", N);
    printf("Sum returned: %20.2f\n", sum);

    float expected = N-1;
    expected = (expected * (expected + 1))/2.0;
    printf("Exepcted answer: %20.2f\n",(expected)); 
    printf("Does output match expected value? %d\n", (sum==expected)?1:0);

    free(input);
    free(output);

    return 0;
}