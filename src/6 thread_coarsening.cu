#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "matrix_utils.h"
#include "timer.h"
#define TILE_SIZE 32
#define COARSE_FACTOR 4

__global__ void mat_mul_kernel(float *A, float *B, float *C, unsigned int N) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x * COARSE_FACTOR + threadIdx.x;

    __shared__ float A_s[TILE_SIZE][TILE_SIZE];
    __shared__ float B_s[TILE_SIZE][TILE_SIZE];

    // Each CUDA thread is now resposible for computing output
    // values for COARSE_FACTOR elements.
    float sum[COARSE_FACTOR];
    for(int i=0; i<COARSE_FACTOR; i++){
        sum[i] = 0.0;
    }

    for(int tile=0; tile< (N+TILE_SIZE-1)/TILE_SIZE; tile++) {

        //Get the tile specific data from A (in DRAM) to A_s (in SRAM)
        if(row < N && ((tile * TILE_SIZE) + threadIdx.x) < N ) {
            A_s[threadIdx.y][threadIdx.x] = A[(row * N) + (tile*TILE_SIZE) + threadIdx.x];
        }
        else {
            A_s[threadIdx.y][threadIdx.x] = 0.0;
        }

        for(int c =0; c < COARSE_FACTOR; c++) {
            // Get the tile specific data from B (in DRAM) to B_s (in SRAM)
            unsigned int B_col = (c * TILE_SIZE) + col;
            if((TILE_SIZE * tile + threadIdx.y) < N && col < N) {
                B_s[threadIdx.y][threadIdx.x] = B[(TILE_SIZE * tile + threadIdx.y) * N + B_col];
            }
            else {
                B_s[threadIdx.y][threadIdx.x] = 0.0;
            }
            __syncthreads();

            // Compute partial sums based on the loaded tile data.
            for(unsigned int j=0; j < TILE_SIZE; j++) {
                sum[c] += A_s[threadIdx.y][j] * B_s[j][threadIdx.x];
            }
            __syncthreads();
        }
    }

    for(int c = 0; c < COARSE_FACTOR; c++)
    {
        unsigned int c_col = (c * TILE_SIZE) + col;
        if (row < N && c_col < N) {
            C[(row * N) + c_col] = sum[c];
        }
    }
    
}

void mat_mul_wrapper(float * A, float * B, float * C, unsigned int N) {
    // Allocate memory on the GPU device.
    float *A_d, *B_d, *C_d;
    size_t mat_size = sizeof(float) * N * N;

    Timer timer;
    timer = initTimer(1);
    startTimer(&timer);

    hipError_t error;
    error = hipMalloc((void **) &A_d, mat_size);
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for matrix A.");
        return;
    }
    error = hipMalloc((void **) &B_d, mat_size);
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for matrix B.");
        return;
    }
    error = hipMalloc((void **) &C_d, mat_size);
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for matrix C.");
        return;
    }
    hipDeviceSynchronize();
    printf("Allocated required memory on the CUDA device.\n\n");
    stopAndPrintElapsed(&timer, "GPU Device Memory Allocation Time: ", CYAN);

    

    // Copy data from Host to GPU.
    timer = initTimer(1);
    startTimer(&timer);
    
    hipMemcpy(A_d, A, mat_size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, mat_size, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "Time To Copy Data to GPU DRAM: ", CYAN);


    // Do the computation on the device.
    timer = initTimer(1);
    startTimer(&timer);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((N + (threadsPerBlock.x * COARSE_FACTOR) - 1) / (threadsPerBlock.x * COARSE_FACTOR), (N + threadsPerBlock.y - 1) / (threadsPerBlock.y)); 
    mat_mul_kernel <<<numBlocks, threadsPerBlock>>>(A_d, B_d, C_d, N);

    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "CUDA Kernel Execution Time: ", GREEN);

    // Copy the results back from GPU  to Host.
    timer = initTimer(1);
    startTimer(&timer);

    hipMemcpy(C, C_d, mat_size, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "Time to COPY Results from GPU to HOST: ", CYAN);

    // Deallocate memory from the GPU device.
    hipFree((void*) A_d);
    hipFree((void*) B_d);
    hipFree((void*) C_d);
    hipDeviceSynchronize();

}

int main(int argc, char** argv) {
    
    //unsigned int N = 40000;
    unsigned int N = 10240;

    size_t mat_size = sizeof(float) * N * N;

    // Create 2 matrixs:
    // A - N*N matrix with Random values
    // B - N*N Identity matrix.
    Matrix A = random_matrix_2D(N , N);
    Matrix B = identity_matrix_2D(N);

    printf("Initialized matrix A and B\n");
    // Allocate memory on the host for holding resulting matrix.
    float * C = (float*) malloc(mat_size);

    // Convert the image to grey;
    mat_mul_wrapper(A.buffer, B.buffer, C, N);
    hipDeviceSynchronize();


    // Manually analyze the first few elements of the matrix to compare results
    int limit = (N < 1024)? N : 1024;
    printf("\nResults of Matrix Multiplication: \n");
    for(int i =0; i<limit; i++) {
      printf("%d: %20.6f    %20.6f\n", i, A.buffer[i], C[i]);
    }

    // Free up allocated space on the host Matrix A, B and buffer holding the results.
    release_matrix(&A);
    release_matrix(&B);
    free(C);

    return 0;
}
