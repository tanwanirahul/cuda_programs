#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "matrix_utils.h"
#include "timer.h"

#define BLOCK_SIZE 1024

__global__ void mat_vec_mul_kernel(SparseMatrix2DCSR mat, float *vector, float *result) {

    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < mat.numRows) {
        float sum = 0.0; 
        for(unsigned int col=mat.rowPtrs[i]; col<mat.rowPtrs[i+1]; col++) {
            float value = mat.values[col];
            sum+= value *  vector[mat.colIdxs[col]];
        }
        result[i] = sum;
    }
}

void mat_vec_mul_wrapper(SparseMatrix2DCSR mat, float *vector, float *result) {
    // Allocate memory on the GPU device.
    float *vector_d, *result_d;
    SparseMatrix2DCSR csr_matrix_d;

    csr_matrix_d.numRows = mat.numRows;
    csr_matrix_d.numCols = mat.numCols;
    csr_matrix_d.numNonZeros = mat.numNonZeros;

    Timer timer;
    timer = initTimer(1);
    startTimer(&timer);

    hipError_t error;
    error = hipMalloc((void **) &vector_d, mat.numCols * sizeof(float));
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for vector.");
        return;
    }
    error = hipMalloc((void **) &result_d, mat.numCols * sizeof(float));
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for result.");
        return;
    }
    error = hipMalloc((void **) &csr_matrix_d.values, mat.numNonZeros * sizeof(float));
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for Sparse matrix values.");
        return;
    }
    error = hipMalloc((void **) &csr_matrix_d.rowPtrs, (mat.numRows + 1) * sizeof(unsigned int));
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for Sparse matrix RowPtrs.");
        return;
    }
    error = hipMalloc((void **) &csr_matrix_d.colIdxs, mat.numNonZeros * sizeof(unsigned int));
    if (error != hipSuccess) {
        printf("\nfailed to allocated memory on CUDA device for Sparse matrix ColIdxs.");
        return;
    }
    hipDeviceSynchronize();
    printf("Allocated required memory on the CUDA device.\n\n");
    stopAndPrintElapsed(&timer, "GPU Device Memory Allocation Time: ", CYAN);

    

    // Copy data from Host to GPU.
    timer = initTimer(1);
    startTimer(&timer);

    // Copy data from host to device.
    hipMemcpy(vector_d, vector, mat.numCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result_d, result, mat.numCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(csr_matrix_d.values, mat.values, mat.numNonZeros * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(csr_matrix_d.rowPtrs, mat.rowPtrs, (mat.numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csr_matrix_d.colIdxs, mat.colIdxs, mat.numNonZeros * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "Time To Copy Data to GPU DRAM: ", CYAN);


    // Do the computation on the device.
    timer = initTimer(1);
    startTimer(&timer);

    unsigned int threadsPerBlock = BLOCK_SIZE;
    unsigned int numBlocks = (mat.numRows + threadsPerBlock - 1) / threadsPerBlock;
    mat_vec_mul_kernel<<<numBlocks, threadsPerBlock>>>(csr_matrix_d, vector_d, result_d);

    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "CUDA Kernel Execution Time: ", GREEN);

    // Copy the results back from GPU  to Host.
    timer = initTimer(1);
    startTimer(&timer);
    hipMemcpy(result, result_d, mat.numRows *sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopAndPrintElapsed(&timer, "Time to COPY Results from GPU to HOST: ", CYAN);

    // Deallocate memory from the GPU device.
    hipFree((void*) vector_d);
    hipFree((void*) result_d);
    hipFree((void*) csr_matrix_d.values);
    hipFree((void*) csr_matrix_d.rowPtrs);
    hipFree((void*) csr_matrix_d.colIdxs);
    hipDeviceSynchronize();

}

int main(int argc, char** argv) {
    
    //unsigned int N = 40000;
    unsigned int N = 10240;

    //Create a sparse matrix and a 1D matrix as a vector:
    SparseMatrix2DCSR A = identity_sparse_matrix_2D_CSR(N);
    Matrix1D b = random_matrix_1D(A.numCols);

    // Allocate memory on the host for holding resulting matrix.
    float * result = (float*) malloc(A.numRows * sizeof(float));

    // Run the matrix vector multiplication.
    mat_vec_mul_wrapper(A, b.buffer, result);
    hipDeviceSynchronize();


    Matrix1D result_mat;
    result_mat.length = A.numRows;
    result_mat.buffer = result;

    printf("\nDoes output match expected results? %d\n", are_matrix_equal_1D(&b, &result_mat));
    
    // Manually analyze the first few rows of the result array.
    int limit = (N < 256)? N : 256;
    printf("\nResults of Matrix Vector Multiplication: \n");
    for(int i =0; i<limit; i++) {
      printf("%d: %20.6f    %20.6f\n", i, b.buffer[i], result_mat.buffer[i]);
    }

    // Free up allocated space on the host Matrix A, B and buffer holding the results.
    release_sparse_matrix_2D_CSR(&A);
    release_matrix_1D(&b);
    free(result);

    return 0;
}